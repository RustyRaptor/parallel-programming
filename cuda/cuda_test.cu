// example cuda program
//
// Created by Ziad Arafat

#include <iostream>
#include <hip/hip_runtime.h>


#include <stdlib.h>

using namespace std;

// kernel function to add up the elements of a vector using reduce algorithm
__global__ void reduce(int *g_idata, int *g_odata)
{
        extern __shared__ int sdata[];

        // each thread loads one element from global to shared mem
        unsigned int tid = threadIdx.x;
        unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
        sdata[tid] = g_idata[i];
        __syncthreads();

        // do reduction in shared mem
        for (unsigned int s = 1; s < blockDim.x; s *= 2)
        {
                if (tid % (2 * s) == 0)
                {
                        sdata[tid] += sdata[tid + s];
                }
                __syncthreads();
        }

        // write result for this block to global mem
        if (tid == 0)
                g_odata[blockIdx.x] = sdata[0];
}

__global__ void vector_add(int *a, int *b, int *c, int N)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N)
        {
                c[i] = a[i] + b[i];
        }
}

int main(int argc, char const *argv[])
{
        // size of the vector
        int N = pow(2, 11);

        // size of the vector in bytes
        size_t bytes = N * sizeof(int);

        // host vectors
        int *h_a, *h_b;

        // allocate memory for host vectors
        h_a = (int *)malloc(bytes);
        h_b = (int *)malloc(bytes);

        // initialize host vectors
        for (int i = 0; i < N; i++)
        {
                // generate a random number
                srand(1); // set the seed to current time
                h_a[i] = rand() % 1000;
                // cout << h_a[i] << " ";
        }

        // device vectors
        int *d_a, *d_b;

        // allocate memory for device vectors
        hipMalloc(&d_a, bytes);
        hipMalloc(&d_b, bytes);

        // copy data from host to device
        hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);

        // number of threads per block
        int NUM_THREADS = 1024;

        // number of blocks
        int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

        // number of bytes for shared memory
        int SHMEM_BYTES = NUM_THREADS * sizeof(int);

        // execute kernel
        reduce<<<NUM_BLOCKS, NUM_THREADS, SHMEM_BYTES>>>(d_a, d_b);

        // copy data from device to host
        hipMemcpy(h_b, d_b, bytes, hipMemcpyDeviceToHost);

        // print the result
        cout << "The sum is " << h_b[0] << endl;

        // test if result is correct
        int sum = 0;
        for (int i = 0; i < N; i++)
        {
                sum += h_a[i];
        }
        cout << "The sum is " << sum << endl;

        // free device memory
        hipFree(d_a);
        hipFree(d_b);

        // free host memory
        free(h_a);
        free(h_b);



        return 0;
}
