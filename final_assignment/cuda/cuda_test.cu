// cuda program to count the number of threes in an array.
// Uses all the threads in a block to count the number of threes in a block.

#define NUM_BLOCKS 1
#define NUM_THREADS 1024

#include <iostream>
#include <hip/hip_runtime.h>


#include <stdlib.h>

using namespace std;
void printPerformanceMetrics(const std::string& metric, double value) {
    std::cout << "\"" << metric << "\": " << value << std::endl;
}
__global__ void count_threes(int *data, int *count, int N)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int local_count = 0;

        if (i < N) {
                local_count = (data[i] == 3) ? 1 : 0;
        }

        extern __shared__ int sdata[];
        sdata[threadIdx.x] = local_count;
        __syncthreads();

        // Perform reduction in shared memory
        for (unsigned int s = 1; s < blockDim.x; s *= 2) {
                int index = 2 * s * threadIdx.x;
                if (index < blockDim.x) {
                        sdata[index] += sdata[index + s];
                }
                __syncthreads();
        }

        // Write result for this block to global memory
        if (threadIdx.x == 0) {
                count[blockIdx.x] = sdata[0];
        }
}

int main(int argc, char const *argv[])
{
        // Parse command line arguments
        if (argc != 2)
        {
                cout << "Usage: " << argv[0] << " N" << endl;
                return 1;
        }
        int N = atoi(argv[1]);

        size_t free_available_gpu_memory, total_available_gpu_memory;

        hipError_t err = hipMemGetInfo(
                &free_available_gpu_memory,
                &total_available_gpu_memory
        );

        if (err != hipSuccess ) {
                cout << hipGetErrorString(err) 
                        << " in " << __FILE__ << " at line " 
                        << __LINE__ << endl;
        }

        // create cuda events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Allocate host memory
        int *h_a, *h_b, *h_c;
        h_a = new int[N];
        h_b = new int[N];
        h_c = new int[N];

        // Initialize host arrays
        for (int i = 0; i < N; i++)
        {
                h_a[i] = rand() % 4;
        }

        // Allocate device memory
        int *d_a, *d_b;
        hipMalloc((void **)&d_a, N * sizeof(int));
        hipMalloc((void **)&d_b, N * sizeof(int));

        // Copy data from host to device memory
        hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);


        // record start time
        hipEventRecord(start);

        // Launch kernel
        int total_count = 0;

        for (int i = 0; i < NUM_BLOCKS; i++) {
                count_threes<<<NUM_BLOCKS, NUM_THREADS, NUM_THREADS * sizeof(int)>>>(d_a, d_b, N);
                hipMemcpy(h_b, d_b, N * sizeof(int), hipMemcpyDeviceToHost);
                total_count += h_b[i];
        }

        // record end time
        hipEventRecord(stop);

        // wait for the stop event to complete
        hipEventSynchronize(stop);

        // calculate elapsed time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // destroy cuda events
        hipEventDestroy(start);
        hipEventDestroy(stop);

        // free memory
        hipFree(d_a);
        hipFree(d_b);
        delete[] h_a;
        delete[] h_b;
        delete[] h_c;
        
        // convert milliseconds to nano seconds
        milliseconds *= 1000000;

        // cout << "Total number of threes in the array: " << total_count << endl;
        printPerformanceMetrics("executionTime", milliseconds);
}
